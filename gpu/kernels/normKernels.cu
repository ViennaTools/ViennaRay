#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vcVectorType.hpp>

extern "C" __global__ void normalize_surface_Triangle_f(
    float *data, const viennacore::Vec3Df *vertex,
    const viennacore::Vec3D<unsigned> *index, const unsigned int numTriangles,
    const float sourceArea, const size_t numRays, const int numData) {
  using namespace viennacore;
  unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int stride = blockDim.x * gridDim.x;

  for (; tidx < numTriangles * numData; tidx += stride) {
    auto elIdx = index[tidx % numTriangles];
    const auto &A = vertex[elIdx[0]];
    const auto &B = vertex[elIdx[1]];
    const auto &C = vertex[elIdx[2]];
    const auto area = Norm(CrossProduct(B - A, C - A)) / 2.f;
    if (area > 1e-6f)
      data[tidx] *= sourceArea / (area * (float)numRays);
    else
      data[tidx] = 0.f;
  }
}

extern "C" __global__ void normalize_surface_Triangle_d(
    double *data, const viennacore::Vec3Df *vertex,
    const viennacore::Vec3D<unsigned> *index, const unsigned int numTriangles,
    const double sourceArea, const size_t numRays, const int numData) {
  using namespace viennacore;
  unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int stride = blockDim.x * gridDim.x;

  for (; tidx < numTriangles * numData; tidx += stride) {
    auto elIdx = index[tidx % numTriangles];
    const auto &A = vertex[elIdx[0]];
    const auto &B = vertex[elIdx[1]];
    const auto &C = vertex[elIdx[2]];
    const double area = Norm(CrossProduct(B - A, C - A)) / 2.;
    if (area > 1e-8)
      data[tidx] *= sourceArea / (area * (double)numRays);
    else
      data[tidx] = 0.;
  }
}

// Areas precomputed on the CPU
extern "C" __global__ void normalize_surface_Disk_f(float *data, float *areas,
                                                    const unsigned int numDisks,
                                                    float sourceArea,
                                                    const size_t numRays,
                                                    const int numData) {
  unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int stride = blockDim.x * gridDim.x;

  for (; tidx < numDisks * numData; tidx += stride) {
    float area = areas[tidx % numDisks];

    if (area > 1e-5f)
      data[tidx] *= sourceArea / (area * (float)numRays);
    else
      data[tidx] = 0.f;
  }
}

// Areas precomputed on the CPU
extern "C" __global__ void normalize_surface_Line_f(float *data, float *areas,
                                                    const unsigned int numLines,
                                                    float sourceArea,
                                                    const size_t numRays,
                                                    const int numData) {
  unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int stride = blockDim.x * gridDim.x;

  for (; tidx < numLines * numData; tidx += stride) {
    float area = areas[tidx % numLines];

    // data[tidx] = area;
    if (area > 1e-5f)
      data[tidx] *= sourceArea / (area * (float)numRays);
    else
      data[tidx] = 0.f;
  }
}